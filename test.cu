#include "hip/hip_runtime.h"
#include <time.h>
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "tensorUtil.h"
#include "trtUtil.h"

clock_t start, end;
Tensor *t, *tcuda;
float *tcuda_data;

int ndim = 3;
int dims[] = {3, 2, 3};
float data[] = {0.0, 1.0, 2.0, 3.0,
                4.0, 5.0, 6.0, 7.0,
                8.0, 9.0, 10.0, 11.0,
                12.0, 13.0, 14.0, 15.0,
                16.0, 17.0};
/*  data[] = {0.0, 2.0, 1.0, */
/*                 5.0, 4.0, 3.0, */
/*                 6.0, 7.0, 6.0, */
/*                 9.0, 10.0, 11.0, */
/*                 12.0, 12.0, 14.0, */
/*                 15.0, 16.0, 16.0}; */


void init()
{
     /* int ndim = 4; */
     /* int dims[] = {20, 24, 78, 72}; */
     /* int dims[] = {20, 24, 1872, 3}; */
     /* size_t data_size = 2695680; */
     /* float *data = (float *)malloc(sizeof(float) * data_size); */
     /* int i; */
     /* for (i = 0; i < data_size; i++) */
     /*      data[i] = 1.0; */

     t = createTensor(data, ndim, dims);
     /* tcuda_data = (float *)cloneMem(t->data, sizeof(float) * t->len, H2D); */
     tcuda = cloneTensor(t, H2D);
     /* tcuda = createTensor(tcuda_data, t->ndim, t->dims); */
     printTensor(t, "%.2f");
}

void testSliceTensor()
{
     /* Tensor *st = createSlicedTensor(t, 2, 2, 1800); */
     Tensor *st = createSlicedTensor(t, 2, 1, 2);
     start = clock();
     /* sliceTensor(t, st, 2, 2, 1800); */
     sliceTensor(t, st, 2, 1, 2);
     end = clock();
     printf("sliceTensor in %ld\n", end - start);
     printTensor(st, "%.2f");

     /* Tensor *stcuda = creatSlicedTensorCuda(tcuda, 2, 2, 1800); */
     Tensor *stcuda = creatSlicedTensorCuda(tcuda, 2, 1, 2);
     start = clock();
     /* sliceTensorCuda(tcuda, stcuda, 2, 2, 1800); */
     sliceTensorCuda(tcuda, stcuda, 2, 1, 2);
     end = clock();
     printf("sliceTensorCuda in %ld\n", end - start);
     float *sthost_data = (float *)cloneMem(stcuda->data, stcuda->len * sizeof(float), D2H);
     Tensor *sthost = createTensor(sthost_data, stcuda->ndim, stcuda->dims);
     printTensor(sthost, "%.2f");
}

void testReshapeTensor()
{
     /* printTensor(t, "%.2f"); */

     /* int newNdim = 3; */
     /* int newDims[] = {3, 3, 2}; */
     int newNdim = 2;
     int newDims[] = {3, 6};
     start = clock();
     Tensor *rt = reshapeTensor(t, newNdim, newDims);
     end = clock();
     printf("reshapeTensor in %ld\n", end - start);
     printTensor(rt, "%.2f");
}

void testReduceArgMax()
{
     assert(isTensorValid(tcuda));
     /* printTensor(t, "%.2f"); */
     Tensor *dst = createReducedTensor(tcuda, tcuda->ndim-2);
     Tensor *arg = createReducedTensor(tcuda, tcuda->ndim-2);
     start = clock();
     reduceArgMax(tcuda, dst, arg, tcuda->ndim-2);
     end = clock();
     printf("reduceArgMax in %ld\n", end - start);

     float *dst_host_data = (float *)cloneMem(dst->data, sizeof(float) * dst->len, D2H);
     Tensor *dst_host = createTensor(dst_host_data, dst->ndim, dst->dims);
     printTensor(dst_host, "%.2f");
     float *arg_host_data = (float *)cloneMem(arg->data, sizeof(float) * arg->len, D2H);
     Tensor *arg_host = createTensor(arg_host_data, arg->ndim, arg->dims);
     printTensor(arg_host, "%.2f");
}

void testMultiplyElement()
{
     float *dst_cuda_data;
     hipMalloc(&dst_cuda_data, sizeof(float) * tcuda->len);
     Tensor *dst = createTensor(dst_cuda_data, tcuda->ndim, tcuda->dims);
     Tensor * src1 = createTensor(tcuda->data, tcuda->ndim, tcuda->dims);
     Tensor * src2 = createTensor(tcuda->data, tcuda->ndim, tcuda->dims);

     start = clock();
     multiplyElement(src1, src2, dst);
     end = clock();
     printf("multiplyElement in %ld\n", end - start);

     float *dst_host_data = (float *)cloneMem(dst->data, sizeof(float) * dst->len, D2H);
     Tensor *dst_host = createTensor(dst_host_data, dst->ndim, dst->dims);
     printTensor(dst_host, "%.2f");
}

void testTransformBboxSQD()
{
     float *delta_host_data = (float *)malloc(sizeof(float) * 24);
     for (int i = 0; i < 24; i++)
          delta_host_data[i] = i * 0.1;
     float *anchor_host_data = (float *)cloneMem(delta_host_data, sizeof(float) * 24, H2H);
     float *delta_cuda_data = (float *)cloneMem(delta_host_data, sizeof(float) * 24, H2D);
     float *anchor_cuda_data = (float *)cloneMem(anchor_host_data, sizeof(float) * 24, H2D);
     float *res_cuda_data;
     hipMalloc(&res_cuda_data, sizeof(float) * 24);

     int dims[] = {1, 4, 6};
     Tensor *delta_host = createTensor(delta_host_data, 3, dims);
     Tensor *anchor_host = createTensor(anchor_host_data, 3, dims);
     Tensor *delta_cuda = createTensor(delta_cuda_data, 3, dims);
     Tensor *anchor_cuda = createTensor(anchor_cuda_data, 3, dims);
     Tensor *res_cuda = createTensor(res_cuda_data, 3, dims);
     float x_scales[] = {1.}, y_scales[] = {1.};
     float *x_scales_device = (float *)cloneMem(x_scales, sizeof(float), H2D);
     float *y_scales_device = (float *)cloneMem(y_scales, sizeof(float), H2D);

     printf("delta_host:\n");
     printTensor(delta_host, "%.6f");
     printf("anchor_host:\n");
     printTensor(anchor_host, "%.6f");
     start =clock();
     transformBboxSQD(delta_cuda, anchor_cuda, res_cuda, 1248, 384, x_scales_device, y_scales_device);
     end = clock();
     printf("transformBboxSQD in %ld\n", end - start);
     float *res_host_data = (float *)cloneMem(res_cuda_data, sizeof(float) * 24, D2H);
     Tensor *res_host = createTensor(res_host_data, 3, dims);
     printTensor(res_host, "%.6f");
}

void testAnchor()
{
     /* int H = 24, W = 78, B = 9; */
     int H = 4, W = 2, B = 9;
     int width = 1248, height = 384;
     float anchor_shape[] = {36, 37, 366, 174, 115, 59, /* w x h, 2 elements one group*/
                             162, 87, 38, 90, 258, 173,
                             224, 108, 78, 170, 72, 43};
     float center_x[W], center_y[H];
     float anchors[H*W*B*4];
     /* int i, j, k; */
     int i;
     for (i = 1; i <= W; i++)
          center_x[i-1] = i * width / (W + 1.0);
     for (i = 1; i <= H; i++)
          center_y[i-1] = i * height / (H + 1.0);
     /* int anchors_dims[] = {W, H, B, 4}; */
     int anchors_dims[] = {4, B, H, W};
     Tensor *anchor_tensor = createTensor(anchors, 4, anchors_dims);
     int a_vol = B * H * W;
     int b_vol = H * W;
     for (i = 0; i < a_vol; i++) {
          anchors[i] = center_x[i % W];
          anchors[a_vol + i] = center_y[i / W % H];
          anchors[a_vol * 2 + i] = anchor_shape[i / b_vol * 2];
          anchors[a_vol * 3 + i] = anchor_shape[i / b_vol * 2 + 1];
     }
     /* int w_vol = H*B*4; */
     /* int h_vol = B*4; */
     /* int b_vol = 4; */
     /* for (i = 0; i < W; i++) { */
     /*      for (j = 0; j < H; j++) { */
     /*           for (k = 0; k < B; k++) { */
     /*                anchors[i*w_vol+j*h_vol+k*b_vol] = center_x[i]; */
     /*                anchors[i*w_vol+j*h_vol+k*b_vol+1] = center_y[j]; */
     /*                anchors[i*w_vol+j*h_vol+k*b_vol+2] = anchor_shape[k*2]; */
     /*                anchors[i*w_vol+j*h_vol+k*b_vol+3] = anchor_shape[k*2+1]; */
     /*           } */
     /*      } */
     /* } */
     printf("anchor_tensor:\n");
     printTensor(anchor_tensor, "%.2f");
}

void testThrustSort()
{
     int n = 9;
     float f[] = {3.1, 9.2, 7.3, 5.4, 4.5, 0.6, 2.7, 6.8, 1.9};
     int id[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
     float *f_device = (float *)cloneMem(f, n * sizeof(float), H2D);
     int *id_device = (int *)cloneMem(id, n * sizeof(int), H2D);
     thrust::sort_by_key(thrust::device, f_device, f_device + n, id_device);
     float *f_host = (float *)cloneMem(f_device, n * sizeof(float), D2H);
     int *id_host = (int *)cloneMem(id_device, n * sizeof(int), D2H);
     for (int i = 0; i < n; i++) {
          printf("%.2f ", f[i]);
     }
     printf("\n");
     for (int i = 0; i < n; i++) {
          printf("%d ", id[i]);
     }
     printf("\n");
     for (int i = 0; i < n; i++) {
          printf("%.2f ", f_host[i]);
     }
     printf("\n");
     for (int i = 0; i < n; i++) {
          printf("%d ", id_host[i]);
     }
     printf("\n");
}

void findThrustBug()
{
     const int SIZE = 16848;
     FILE *infile = fopen("sort_bug.txt", "r");
     FILE *outfile = fopen("sorted.txt", "w");
     float array[SIZE];
     float *array_device;
     int id[SIZE];
     int *id_device;
     int *id_host;

     for (int i = 0; i < SIZE; i++) {
          fscanf(infile, "%f", &array[i]);
          id[i] = i;
     }
     hipMalloc(&array_device, SIZE * sizeof(float));
     hipMemcpy(array_device, array, SIZE * sizeof(float), hipMemcpyHostToDevice);
     hipMalloc(&id_device, SIZE * sizeof(int));
     hipMemcpy(id_device, id, SIZE * sizeof(int), hipMemcpyHostToDevice);
     thrust::sort_by_key(thrust::device, array_device, array_device + SIZE, id_device);
     id_host = (int *)malloc(SIZE * sizeof(int));
     hipMemcpy(id_host, id_device, SIZE * sizeof(int), hipMemcpyDeviceToHost);
     for (int i = 0; i < SIZE; i++)
          fprintf(outfile, "%d\n", id_host[i]);
     fclose(infile);
     fclose(outfile);
}

void testOpencv()
{
     std::vector<std::string> imglist = getImageList("data/example");
     /* cv::Mat img = readImage(imglist.begin(), 1248, 768); */
     std::string filename = imglist[0];
     printf("filename: %s\n", filename.c_str());

     cv::Mat img = cv::imread(filename);
     printf("img.total(): %ld\n", img.total());
     cv::resize(img, img, cv::Size(1248, 384));
}

void testIou()
{
     float bbox0[] = {0, 0, 0, 0};
     float bbox1[] = {0, 0, 0, 0};
     printf("%f\n", computeIou(bbox0, bbox1));
}

void testPickElements()
{
     float src_host[] = {0.0, 1.0, 2.0, 3.0,
                       4.0, 5.0, 6.0, 7.0,
                       8.0, 9.0, 10.0, 11.0,
                       12.0, 13.0, 14.0, 15.0,
                       16.0, 17.0};
     int index[] = {9, 15, 2, 17, 11};
     int len = 5;
     int *index_device = (int *)cloneMem(index, len * sizeof(int), H2D); /* remember this! */
     float *src_device = (float *)cloneMem(src_host, 18 * sizeof(float), H2D);
     float *dst_device;
     hipMalloc(&dst_device, len * sizeof(float));

     pickElements(src_device, dst_device, 1, index_device, len);

     float *dst_host = (float *)cloneMem(dst_device, len * sizeof(float), D2H);
     for (int i = 0; i < 18; i++)
          printf("%.2f ", src_host[i]);
     printf("\n");
     for (int i = 0; i < len; i++)
          printf("%.2f ", dst_host[i]);
     printf("\n");
}

int main(int argc, char *argv[])
{
     init();
     /* testSliceTensor(); */
     /* testReshapeTensor(); */
     /* testReduceArgMax(); */
     /* testMultiplyElement(); */
     testTransformBboxSQD();
     /* testAnchor(); */
     /* testThrustSort(); */
     /* findThrustBug(); */
     /* testOpencv(); */
     /* testIou(); */
     /* testPickElements(); */
}
