#include "errorHandle.h"

void checkError(hipError_t error)
{
     if (error == hipSuccess)
          return;
     fprintf(stderr, "CUDA_ERROR(%s): %s\n", hipGetErrorName(error), hipGetErrorString(error));
     abort();
}
