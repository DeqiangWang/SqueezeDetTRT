#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

static __device__ float E = 2.718281828;

static __device__ int getIndex(int *ids, int ndim, int *dims)
{
     int i, id;
     for (i = 0, id = ids[0]; i < ndim-1; i++)
          id = dims[i+1] * id + ids[i+1];
     return id;
}

static __device__ void getIndexes(int id, int *ids, int ndim, int *dims)
{
     for (int i = ndim-1; i >=0; i--) {
          ids[i] = id % dims[i];
          id = id / dims[i];
     }
}

/* __global__ void sliceTensorKernel(float *src, float *dst, int sdim, int ddim, int start, int block_size) */
/* { */
/*      int di = blockIdx.x * block_size + threadIdx.x; */
/*      /\* si is the index of src elements to be copied. */
/*         The "block index" of src[si] is (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) *\/ */
/*      int si = (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) * block_size + threadIdx.x; */
/*      dst[di] = src[si]; */
/* } */

__global__ void sliceTensorKernel(float *src, float *dst, int start, int s_vol, int d_vol, int vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = di / d_vol * s_vol + di % d_vol + start * vol;
     dst[di] = src[si];
}

__global__ void reduceArgMaxKernel(float *src, float *dst, float *arg, int dim_size, int reduce_vol, int batch_vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     /* src[si] is the first element in this thread to be compared, then
        si = batch_vol * batch + (di - reduce_vol * batch),
        where batch = di / reduce_vol,
        which is the same as the following code: */
     int si = (batch_vol - reduce_vol) * (di / reduce_vol) + di;
     float now = src[si], max = now;
     int maxi = 0;
     for (int i = 1; i < dim_size; i++) {
          now = src[si+i*reduce_vol];
          if (now > max) {
               max = now;
               maxi = i;
          }
     }
     dst[di] = max;
     arg[di] = maxi;
}

__global__ void multiplyElementKernel(float *src1, float *src2, float *dst, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     dst[di] = src1[di] * src2[di];
}

__global__ void transposeTensorKernel(float *src, float *dst, int ndim, int *s_dims, int *d_dims, int *s_ids, int *d_ids, int *axes, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     int *t_s_ids = s_ids + di * ndim;
     int *t_d_ids = d_ids + di * ndim;
     getIndexes(di, t_d_ids, ndim, d_dims);
     for (int i = 0; i < ndim; i++)
          t_s_ids[axes[i]] = t_d_ids[i];
     int si = getIndex(t_s_ids, ndim, s_dims);

     dst[di] = src[si];
}

__global__ void transformBboxSQDKernel(float *delta, float *anchor, float *res, float width, float height, float *x_scales, float *y_scales, int anchor_num, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     int batch_idx = di / anchor_num;
     float x_scale = x_scales[batch_idx];
     float y_scale = y_scales[batch_idx];
     float img_width = width / x_scale;
     float img_height = height / y_scale;

     /* si is the index of the first elements to be computed in the thread, then
        si = 4 * anchor_num * batch_idx + (di - anchor_num * batch_idx),
        which is the same as the following code: */
     int si = 3 * anchor_num * batch_idx  + di;
     /* take 4 elements from each of delta and anchor */
     float d[4] = {delta[si], delta[si+anchor_num], delta[si+2*anchor_num], delta[si+3*anchor_num]};
     float a[4] = {anchor[si], anchor[si+anchor_num], anchor[si+2*anchor_num], anchor[si+3*anchor_num]};
     /* compute and put 4 result elements to res, according to SqueezeDet's source code */
     float cx = a[0] + d[0] * a[2] / x_scale;
     float cy = a[1] + d[1] * a[3] / y_scale;
     float w = a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E) / x_scale;
     float h = a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E) / y_scale;
     res[di] = min(max(cx - w * 0.5, 0), img_width - 1);
     res[di+anchor_num] = min(max(cy - h * 0.5, 0), img_height - 1);
     res[di+2*anchor_num] = max(min(cx + w * 0.5, img_width - 1), 0);
     res[di+3*anchor_num] = max(min(cy + h * 0.5, img_height - 1), 0);
}

__global__ void pickElementsKernel(float *src, float *dst, int *idx, int stride, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = idx[di];
     for (int i = 0; i < stride; i++)
          dst[di*stride+i] = src[si*stride+i];
}
