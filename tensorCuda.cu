#include <hip/hip_runtime.h>

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

static __device__ float E = 2.718281828;

/* __global__ void sliceTensorKernel(float *src, float *dst, int sdim, int ddim, int start, int block_size) */
/* { */
/*      int di = blockIdx.x * block_size + threadIdx.x; */
/*      /\* si is the index of src elements to be copied. */
/*         The "block index" of src[si] is (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) *\/ */
/*      int si = (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) * block_size + threadIdx.x; */
/*      dst[di] = src[si]; */
/* } */

__global__ void sliceTensorKernel(float *src, float *dst, int start, int s_vol, int d_vol, int vol, int block_size)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     int si = di / d_vol * s_vol + di % d_vol + start * vol;
     dst[di] = src[si];
}

__global__ void reduceArgMaxKernel(float *src, float *dst, float *arg, int dim_size, int reduce_vol, int batch_vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     /* src[si] is the first element in this thread to be compared, then
        si = batch_vol * batch + (di - reduce_vol * batch),
        where batch = di / reduce_vol,
        which is the same as the following code: */
     int si = (batch_vol - reduce_vol) * (di / reduce_vol) + di;
     float now = src[si], max = now;
     int maxi = 0;
     for (int i = 1; i < dim_size; i++) {
          now = src[si+i*reduce_vol];
          if (now > max) {
               max = now;
               maxi = i;
          }
     }
     dst[di] = max;
     arg[di] = maxi;
}

__global__ void multiplyElementKernel(float *src1, float *src2, float *dst, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     dst[di] = src1[di] * src2[di];
}

__global__ void transformBboxSQDKernel(float *delta, float *anchor, float *res, float width, float height, float *x_scales, float *y_scales, int anchor_num, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     int batch_idx = di / anchor_num;
     float x_scale = x_scales[batch_idx];
     float y_scale = y_scales[batch_idx];
     float img_width = width / x_scale;
     float img_height = height / y_scale;

     /* si is the index of the first elements to be computed in the thread, then
        si = 4 * anchor_num * batch_idx + (di - anchor_num * batch_idx),
        which is the same as the following code: */
     int si = 3 * anchor_num * batch_idx  + di;
     /* take 4 elements from each of delta and anchor */
     float d[4] = {delta[si], delta[si+anchor_num], delta[si+2*anchor_num], delta[si+3*anchor_num]};
     float a[4] = {anchor[si], anchor[si+anchor_num], anchor[si+2*anchor_num], anchor[si+3*anchor_num]};
     /* compute and put 4 result elements to res, according to SqueezeDet's source code */
     float cx = a[0] + d[0] * a[2] / x_scale;
     float cy = a[1] + d[1] * a[3] / y_scale;
     float w = a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E) / x_scale;
     float h = a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E) / y_scale;
     res[di] = min(max(cx - w * 0.5, 0), img_width - 1);
     res[di+anchor_num] = min(max(cy - h * 0.5, 0), img_height - 1);
     res[di+2*anchor_num] = max(min(cx + w * 0.5, img_width - 1), 0);
     res[di+3*anchor_num] = max(min(cy + h * 0.5, img_height - 1), 0);
}

__global__ void pickElementsKernel(float *src, float *dst, int *idx, int stride, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = idx[di];
     for (int i = 0; i < stride; i++)
          dst[di*stride+i] = src[si*stride+i];
}
