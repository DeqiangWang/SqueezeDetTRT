#include <hip/hip_runtime.h>

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

static __device__ float E = 2.718281828;

__global__ void sliceTensorKernel(float *src, float *dst, int sdim, int ddim, int start, int block_size)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     int si = (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) * block_size + threadIdx.x;
     dst[di] = src[si];
}

__global__ void reduceArgMaxKernel(float *src, float *dst, float *arg, int dim_size, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = di * dim_size;
     float now = src[si], max = now;
     int maxi = 0;
     for (int i = 1; i < dim_size; i++) {
          now = src[si+i];
          if (now > max) {
               max = now;
               maxi = i;
          }
     }
     dst[di] = max;
     arg[di] = maxi;
}

__global__ void multiplyElementKernel(float *src1, float *src2, float *dst, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     dst[di] = src1[di] * src2[di];
}

__global__ void transformBboxSQDKernel(float *delta, float *anchor, float *res, float img_width, float img_height, float x_scale, float y_scale, int block_size, int total)
{
     int di = (blockIdx.x * block_size + threadIdx.x) * 4;
     if (di >= total)
          return;
     float d[4] = {delta[di], delta[di+1], delta[di+2], delta[di+3]};
     float a[4] = {anchor[di], anchor[di+1], anchor[di+2], anchor[di+3]};
     float cx = a[0] + d[0] * a[2] / x_scale;
     float cy = a[1] + d[1] * a[3] / y_scale;
     float w = a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E) / x_scale;
     float h = a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E) / y_scale;
     res[di] = min(max(cx - w * 0.5, 0), img_width - 1);
     res[di+1] = min(max(cy - h * 0.5, 0), img_height - 1);
     res[di+2] = max(min(cx + w * 0.5, img_width - 1), 0);
     res[di+3] = max(min(cy + h * 0.5, img_height - 1), 0);
}

__global__ void pickElementsKernel(float *src, float *dst, int *idx, int len, int stride, int block_size)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= len)
          return;
     int si = idx[di];
     for (int i = 0; i < stride; i++)
          dst[di*stride+i] = src[si*stride+i];
}
