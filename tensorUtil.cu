#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include "tensorCuda.h"
#include "tensorUtil.h"
#include "errorHandle.h"
#include "sdt_alloc.h"

#define MAXDIM 8
#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

static float EPSILON = 1e-16;

static void assertTensor(const Tensor *tensor)
{
     assert(tensor && tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0);
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isTensorValid(const Tensor *tensor)
{
     return (tensor && tensor->data &&
             tensor->ndim < MAXDIM && tensor->ndim > 0 &&
             tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isShapeEqual(const Tensor *t1, const Tensor *t2)
{
     assertTensor(t1);
     assertTensor(t2);
     if (t1->ndim == t2->ndim) {
          int ndim = t1->ndim;
          while (--ndim >= 0)
               if (t1->dims[ndim] != t2->dims[ndim])
                    return 0;
          return 1;
     }
     return 0;
}

/* can only identify host memory alloced by hipHostMalloc, etc */
int isHostMem(const void *ptr)
{
     hipPointerAttribute_t attributes;
     checkError(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeHost;
}

int isDeviceMem(const void *ptr)
{
     hipPointerAttribute_t attributes;
     checkError(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeDevice;
}

void *cloneMem(const void *src, size_t size, CloneKind kind)
{
     assert(src);
     void *p;
     switch (kind) {
     case H2H:
          p = sdt_alloc(size);
          memmove(p, src, size);
          return p;
     case H2D:
          checkError(hipMalloc(&p, size));
          checkError(hipMemcpy(p, src, size, hipMemcpyHostToDevice));
          return p;
     case D2D:
          assert(isDeviceMem(src));
          checkError(hipMalloc(&p, size));
          checkError(hipMemcpy(p, src, size, hipMemcpyDeviceToDevice));
          return p;
     case D2H:
          assert(isDeviceMem(src));
          p = sdt_alloc(size);
          checkError(hipMemcpy(p, src, size, hipMemcpyDeviceToHost));
          return p;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }

}

Tensor *cloneTensor(const Tensor *src, CloneKind kind)
{
     assert(isTensorValid(src));
     float *data = (float *)cloneMem(src->data, src->len * sizeof(float), kind);
     Tensor *dst = createTensor(data, src->ndim, src->dims);
     return dst;
}

void *repeatMem(void *data, size_t size, int times, CloneKind kind)
{
     assert(data && times > 0);
     void *p, *dst;
     int i;
     switch (kind) {
     case H2H:
          dst = p = sdt_alloc(size * times);
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               memmove(p, data, size);
          return dst;
     case H2D:
          checkError(hipMalloc(&p, size * times));
          dst = p;
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyHostToDevice));
          return dst;
     case D2D:
          assert(isDeviceMem(data));
          checkError(hipMalloc(&p, size * times));
          dst = p;
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyDeviceToDevice));
          return dst;
     case D2H:
          assert(isDeviceMem(data));
          dst = p = sdt_alloc(size * times);
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyDeviceToHost));
          return dst;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }
}


int computeLength(int ndim, const int *dims)
{
     if (dims) {
          int i, len = 1;
          for (i = 0; i < ndim; i++)
               len *= dims[i];
          return len;
     }
     fprintf(stderr, "Warning: null dims in computeLength\n");
     return 0;
}

Tensor *createTensor(float *data, int ndim, const int *dims)
{
     Tensor *t = (Tensor *)sdt_alloc(sizeof(Tensor));
     t->data = data;
     t->ndim = ndim;
     t->dims = (int *)sdt_alloc(sizeof(int) * ndim);
     memmove(t->dims, dims, sizeof(int) * ndim);
     t->len = computeLength(ndim, dims);
     return t;
}

Tensor *mallocTensor(int ndim, const int* dims, const MallocKind mkind)
{
     Tensor *t = createTensor(NULL, ndim, dims);
     float *f;

     switch (mkind) {
     case HOST:
          f = (float *)sdt_alloc(t->len * sizeof(float));
          break;
     case DEVICE:
          checkError(hipMalloc(&f, t->len * sizeof(float)));
          break;
     default:
          fprintf(stderr, "unknown MallocKind %d\n", mkind);
          return NULL;
     }

     t->data = f;
     return t;
}

void freeTensor(Tensor *t, int do_free_data)
{
     assert(isTensorValid(t));
     sdt_free(t->dims);
     if (do_free_data) {
          if (isDeviceMem(t->data))
               checkError(hipFree(t->data));
          else
               sdt_free(t->data);
     }
     sdt_free(t);
}

void fprintTensor(FILE *stream, const Tensor *tensor, const char *fmt)
{
     assertTensor(tensor);
     int dim_sizes[MAXDIM], dim_levels[MAXDIM]; /* dimision size and how deep current chars go */
     int ndim = tensor->ndim, len = tensor->len, *dims = tensor->dims; /* pointer short cut */
     float *data = tensor->data;
     char left_buf[MAXDIM+1], right_buf[MAXDIM+1]; /* buffer for brackets */
     char *lp = left_buf, *rp = right_buf;
     size_t right_len;
     int i, j, k;

     dim_sizes[ndim-1] = tensor->dims[ndim-1];
     dim_levels[ndim-1] = 0;
     for (i = ndim-2; i >= 0; i--) {
          dim_sizes[i] = dims[i] * dim_sizes[i+1];
          dim_levels[i] = 0;
     }
     for (i = 0; i < len; i++) {
          for (j = 0; j < ndim; j++) {
               if (i % dim_sizes[j] == 0)
                    dim_levels[j]++;
               if (dim_levels[j] == 1) {
                    *lp++ = '[';
                    dim_levels[j]++;
               }
               if (dim_levels[j] == 3) {
                    *rp++ = ']';
                    if (j != 0 && dim_levels[j] > dim_levels[j-1]) {
                         *lp++ = '[';
                         dim_levels[j] = 2;
                    } else
                         dim_levels[j] = 0;
               }
          }
          *lp = *rp = '\0';
          fprintf(stream, "%s", right_buf);
          if (*right_buf != '\0') {
               fprintf(stream, "\n");
               right_len = strlen(right_buf);
               for (k = ndim-right_len; k > 0; k--)
                    fprintf(stream, " ");
          }
          fprintf(stream, "%s", left_buf);
          if (*left_buf == '\0')
               fprintf(stream, " ");
          fprintf(stream, fmt, data[i]);
          lp = left_buf, rp = right_buf;
     }
     for (j = 0; j < ndim; j++)
          fprintf(stream, "]");
     fprintf(stream, "\n");
}

void printTensor(const Tensor *tensor, const char *fmt)
{
     fprintTensor(stdout, tensor, fmt);
}

void fprintDeviceTensor(FILE *stream, const Tensor *d_tensor, const char *fmt)
{
     assert(isTensorValid(d_tensor));
     Tensor *h_tensor = cloneTensor(d_tensor, D2H);
     fprintTensor(stream, h_tensor, fmt);
     free(h_tensor->data); /* TODO: free t_tensor */
}

void printDeviceTensor(const Tensor *d_tensor, const char *fmt)
{
     fprintDeviceTensor(stdout, d_tensor, fmt);
}

void saveTensor(const char *file_name, const Tensor *tensor, const char *fmt)
{
     FILE *fp = fopen(file_name, "w");
     fprintTensor(fp, tensor, fmt);
     fclose(fp);
}

void saveDeviceTensor(const char *file_name, const Tensor *d_tensor, const char *fmt)
{
     FILE *fp = fopen(file_name, "w");
     fprintDeviceTensor(fp, d_tensor, fmt);
     fclose(fp);
}

/* Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src)); */
/*      assert(dim <= src->ndim && dim >= 0); */
/*      assert(len+start <= src->dims[dim]); */

/*      Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /\* new tensor *\/ */
/*      dst->ndim = src->ndim; */
/*      dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim); */
/*      memmove(dst->dims, src->dims, sizeof(int) * dst->ndim); */
/*      dst->dims[dim] = len; */
/*      dst->len = src->len / src->dims[dim] * len; */
/*      dst->data = (float *)sdt_alloc(dst->len * sizeof(float)); */
/*      return dst; */
/* } */

/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i, block_size, block_num; /\* block size and number for copy operation *\/ */
/*      for (i = dim+1, block_size = 1; i < dst->ndim; i++) */
/*           block_size *= dst->dims[i]; */
/*      for (i = 0, block_num = 1; i <= dim; i++) */
/*           block_num *= dst->dims[i]; */

/*      int index; */
/*      float *dp = dst->data, *sp = src->data; */
/*      size_t floats_size = block_size * sizeof(float); */
/*      for (i = 0; i < block_num; i++) { */
/*           index = i / len * src->dims[dim] + i % len + start; */
/*           memmove(dp+i*block_size, sp+index*block_size, floats_size); */
/*      } */

/*      return dst; */
/* } */

Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len)
{
     assert(isTensorValid(src));
     assert(dim <= MAXDIM);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     checkError(hipMalloc(&dst->data, sizeof(float) * dst->len));
     return dst;
}

/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(isDeviceMem(src->data) && isDeviceMem(dst->data)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i, block_size, block_num; /\* block size and number of cuda threads *\/ */
/*      int ddim = dst->dims[dim], sdim = src->dims[dim]; */
/*      for (i = dim+1, block_size = 1; i < dst->ndim; i++) */
/*           block_size *= dst->dims[i]; */
/*      for (i = 0, block_num = 1; i <= dim; i++) */
/*           block_num *= dst->dims[i]; */

/*      sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, sdim, ddim, start, block_size); */
/*      return dst; */
/* } */

Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len)
{
     assert(isTensorValid(src) && isTensorValid(dst));
     assert(isDeviceMem(src->data) && isDeviceMem(dst->data));
     assert(dst->ndim == src->ndim);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]);

     int i, d_vol, s_vol, vol;
     int thread_num, block_size, block_num; /* block size and number of cuda threads */
     for (i = dim+1, vol = 1; i < dst->ndim; i++)
          vol *= dst->dims[i];
     d_vol = vol * dst->dims[dim];
     s_vol = vol * src->dims[dim];
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, start, s_vol, d_vol, vol, block_size, thread_num);
     return dst;
}

/* in-place reshape tensor */
Tensor *reshapeTensor(const Tensor *src, int newNdim, const int *newDims)
{
     assert(isTensorValid(src));
     assert(newDims);
     assert(src->len == computeLength(newNdim, newDims));
     Tensor *dst = createTensor(src->data, newNdim, newDims); /* new tensor */
     return dst;
}

Tensor *createReducedTensor(const Tensor *src, int dim)
{
     assert(isTensorValid(src));
     assert(dim < src->ndim && dim >= 0);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor));
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = 1;
     dst->len = computeLength(dst->ndim, dst->dims);
     checkError(hipMalloc(&dst->data, sizeof(float) * dst->len));
     return dst;
}

void *reduceArgMax(const Tensor *src, Tensor *dst, Tensor *arg, int dim)
{
     assert(isTensorValid(src) && isTensorValid(dst) && isTensorValid(arg));
     assert(isDeviceMem(src->data) && isDeviceMem(dst->data) && isDeviceMem(arg->data));
     assert(dim < src->ndim && dim >= 0);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == 1 : dst->dims[i] == src->dims[i] &&
                 i == dim ? arg->dims[i] == 1 : arg->dims[i] == src->dims[i]);

     /* suppose the shape of src is [N, C, H, W], dim = 1, then thread_num is N x H x W
        reduce_vol is H x W, index_vol is C x H x W */
     int i, thread_num, block_size, block_num, reduce_vol, index_vol;
     for (i = dim+1, thread_num = 1; i < dst->ndim; i++)
          thread_num *= dst->dims[i];
     reduce_vol = thread_num;
     index_vol = thread_num * src->dims[dim];
     for (i = 0; i < dim; i++)
          thread_num *= dst->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     reduceArgMaxKernel<<<block_num, block_size>>>(src->data, dst->data, arg->data, src->dims[dim], reduce_vol, index_vol, block_size, thread_num);
     return dst;
}

Tensor *multiplyElement(const Tensor *src1, const Tensor *src2, Tensor *dst)
{
     assert(isShapeEqual(src1, src2));
     assert(isShapeEqual(src1, dst));
     assert(isDeviceMem(src1->data) && isDeviceMem(src2->data) && isDeviceMem(dst->data));

     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     multiplyElementKernel<<<block_num, block_size>>>(src1->data, src2->data, dst->data, block_size, dst->len);
     return dst;
}

/* (optional) workspace size equals (sizeof(int) * dst->ndim * dst->len), two of them */
Tensor *transposeTensor(const Tensor *src, Tensor *dst, int *axes, int **workspace)
{
     assert(isTensorValid(src) && isTensorValid(dst));
     assert(src->len == dst->len);
     assert(src->ndim == dst->ndim);

     int *s_ids, *d_ids, *s_dims, *d_dims;
     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;
     s_dims = (int *)cloneMem(src->dims, sizeof(int) * src->ndim, H2D);
     d_dims = (int *)cloneMem(dst->dims, sizeof(int) * dst->ndim, H2D);
     if (!workspace) {
          checkError(hipMalloc(&s_ids, sizeof(int) * dst->ndim * thread_num));
          checkError(hipMalloc(&d_ids, sizeof(int) * dst->ndim * thread_num));
     } else {
          s_ids = workspace[0];
          d_ids = workspace[1];
     }

     transposeTensorKernel<<<block_num, block_size>>>(src->data, dst->data, dst->ndim, s_dims, d_dims, s_ids, d_ids, axes, block_size, thread_num);

     if (!workspace) {
          checkError(hipFree(s_ids));
          checkError(hipFree(d_ids));
     }
     checkError(hipFree(s_dims));
     checkError(hipFree(d_dims));
     return dst;
}

/* transform from bbox delta to bbox coordinates, using hyper param EXP_THRESH = 1.0.
   delta, anchor, res are all of the same shape [..., 4]
   width and height are resized image width and height.
   x_scales and y_scales are (temporary) pointers to width/original_width and height/original_height. */
Tensor *transformBboxSQD(const Tensor *delta, const Tensor *anchor, Tensor *res, float width, float height, float img_width, float img_height)
{
     assert(isShapeEqual(delta, anchor));
     assert(isShapeEqual(delta, res));
     assert(delta->ndim == 5);
     assert(delta->dims[4] == 4);
     assert(isDeviceMem(delta->data) && isDeviceMem(anchor->data) && isDeviceMem(res->data));

     /* take 4 elements from each of delta and anchor,
        and put 4 result elements to res in one thread */
     int i, thread_num, block_size, block_num;
     for (i = 0, thread_num = 1; i < res->ndim-1; i++)
          thread_num *= res->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     transformBboxSQDKernel<<<block_num, block_size>>>(delta->data, anchor->data, res->data, width, height, img_width, img_height, block_size, thread_num);
     return res;
}

void tensorIndexSort(Tensor *src, int *idx)
{
     assert(isTensorValid(src));
     assert(idx);
     assert(isDeviceMem(src->data) && isDeviceMem(idx));

     /* the thrust call below can be unreliable, sometimes produces error */
     /* now it works with compilation flag -arch=sm_35 */
     /* TODO: replace thrust call by our own kernel */
     thrust::sort_by_key(thrust::device, src->data, src->data + src->len, idx, thrust::greater<float>());
}

void pickElements(float *src, float *dst, int stride, int *idx, int len)
{
     assert(src && dst && idx);
     assert(isDeviceMem(src) && isDeviceMem(dst) && isDeviceMem(idx));

     int thread_num, block_size, block_num;
     thread_num = len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     pickElementsKernel<<<block_num, block_size>>>(src, dst, idx, stride, block_size, thread_num);
}

/* void pickElements(float* src,float* dst,int stride,int* idx,int len) */
/* { */
/*      assert(src && dst && idx); */

/*      for (int i = 0; i < len; i++) { */
/*           for (int j = 0; j < stride; j++) { */
/*                fprintf(stderr, "i: %d j: %d idx[i]: %d src[idx[i]]: %.2f", */
/*                        i, j, idx[i], src[idx[i]]); */
/*                fprintf(stderr, "\n"); */
/*                dst[i*stride+j] = src[idx[i]*stride+j]; */
/*           } */
/*      } */
/* } */

/* compute the iou of two bboxes whose elements are {top_left_x, top_left_y, bottom_right_x, bottom_right_y} */
float computeIou(float *bbox0, float *bbox1)
{
     assert(bbox0 && bbox1);

     float lr, tb;              /* left-right, top-bottom for intersection*/
     float intersection, total;
     lr = min(bbox0[2], bbox1[2]) - max(bbox0[0], bbox1[0]);
     if (lr >= 0) {
          tb = min(bbox0[3], bbox1[3]) - max(bbox0[1], bbox1[1]);
          if (tb >= 0) {
               intersection = tb * lr + EPSILON;
               total = (bbox0[2] - bbox0[0]) * (bbox0[3] - bbox0[1]) +
                    (bbox1[2] - bbox1[0]) * (bbox1[3] - bbox1[1]) - intersection;
               return intersection / (total + EPSILON);
          }
     }
     return 0;
}
