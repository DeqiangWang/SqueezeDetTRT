#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
/* #include "tensorCuda.h" */
#include "tensorUtil.h"
#include "errorHandle.h"
#include "sdt_alloc.h"

#define MAXDIM 8
#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))
#define MAX_THREADS_PER_BLOCK 1024
#define BLOCK_SIZE MAX_THREADS_PER_BLOCK

static float EPSILON = 1e-16;

static __device__ float E = 2.718281828;

static __device__ int getIndex(int *ids, int ndim, int *dims)
{
     int i, id;
     for (i = 0, id = ids[0]; i < ndim-1; i++)
          id = dims[i+1] * id + ids[i+1];
     return id;
}

static __device__ void getIndexes(int id, int *ids, int ndim, int *dims)
{
     for (int i = ndim-1; i >=0; i--) {
          ids[i] = id % dims[i];
          id = id / dims[i];
     }
}

/* __global__ void sliceTensorKernel(float *src, float *dst, int sdim, int ddim, int start, int block_size) */
/* { */
/*      int di = blockIdx.x * block_size + threadIdx.x; */
/*      /\* si is the index of src elements to be copied. */
/*         The "block index" of src[si] is (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) *\/ */
/*      int si = (blockIdx.x / ddim * sdim + blockIdx.x % ddim + start) * block_size + threadIdx.x; */
/*      dst[di] = src[si]; */
/* } */

/* __global__ void sliceTensorKernel(float *src, float *dst, int ndim, int *sdims, int *ddims, int naxis, int *axes, int *n_pick_ids, int **pick_ids, int *sids, int *dids, int block_size, int total) */
/* { */
/*      int di = blockIdx.x * block_size + threadIdx.x; */
/*      if (di >= total) */
/*           return; */

/*      int *t_sids = sids + di * ndim; */
/*      int *t_dids = dids + di * ndim; */
/*      getIndexes(di, t_dids, ndim, ddims); */
/*      int i, j; */
/*      for (i = 0, j = 0; i < ndim; i++) { */
/*           if (i != axes[j]) { */
/*                t_sids[i] = t_dids[i]; */
/*           } else { */
/*                t_sids[i] = pick_ids[j][t_dids[i]]; */
/*                j++; */
/*           } */
/*      } */

/*      int si = getIndex(t_sids, ndim, sdims); */
/*      dst[di] = src[si]; */
/* } */

__global__ void sliceTensorKernel(float *src, float *dst, int start, int s_vol, int d_vol, int vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = di / d_vol * s_vol + di % d_vol + start * vol;
     dst[di] = src[si];
}

__global__ void reduceArgMaxKernel(float *src, float *dst, float *arg, int dim_size, int reduce_vol, int batch_vol, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     /* src[si] is the first element in this thread to be compared, then
        si = batch_vol * batch + (di - reduce_vol * batch),
        where batch = di / reduce_vol,
        which is the same as the following code: */
     int si = (batch_vol - reduce_vol) * (di / reduce_vol) + di;
     float now = src[si], max = now;
     int maxi = 0;
     for (int i = 1; i < dim_size; i++) {
          now = src[si+i*reduce_vol];
          if (now > max) {
               max = now;
               maxi = i;
          }
     }
     dst[di] = max;
     arg[di] = maxi;
}

__global__ void multiplyElementKernel(float *src1, float *src2, float *dst, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     dst[di] = src1[di] * src2[di];
}

__global__ void transposeTensorKernel(float *src, float *dst, int ndim, int *s_dims, int *d_dims, int *s_ids, int *d_ids, int *axes, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     int *t_s_ids = s_ids + di * ndim;
     int *t_d_ids = d_ids + di * ndim;
     getIndexes(di, t_d_ids, ndim, d_dims);
     for (int i = 0; i < ndim; i++)
          t_s_ids[axes[i]] = t_d_ids[i];
     int si = getIndex(t_s_ids, ndim, s_dims);

     dst[di] = src[si];
}

__global__ void transformBboxSQDKernel(float *delta, float *anchor, float *res, float width, float height, float img_width, float img_height, int x_shift, int y_shift, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;

     /* int batch_idx = di / anchor_num; */
     /* now only support batch_size = 1 */
     float x_scale = 1.0 * img_width / width;
     float y_scale = 1.0 * img_height / height;

     /* (not used) si is the index of the first elements to be computed in the thread, then
        si = 4 * anchor_num * batch_idx + (di - anchor_num * batch_idx),
        which is the same as the following code: */
     /* int si = 3 * anchor_num * batch_idx  + di; */
     /* take 4 elements from each of delta and anchor */
     int si = di * 4;
     float d[4] = {delta[si], delta[si+1], delta[si+2], delta[si+3]};
     float a[4] = {anchor[si], anchor[si+1], anchor[si+2], anchor[si+3]};
     /* compute and put 4 result elements to res, according to SqueezeDet's source code */

     /* TODO: don't know why (maybe the resize), always has some shift compared to groundtruth*/
     float cx = (a[0] + d[0] * a[2]) * x_scale + x_shift;
     float cy = (a[1] + d[1] * a[3]) * y_scale + y_shift;
     float w = (a[2] * (d[2] < 1 ? expf(d[2]) : d[2] * E)) * x_scale;
     float h = (a[3] * (d[3] < 1 ? expf(d[3]) : d[3] * E)) * y_scale;
     res[si] = min(max(cx - w * 0.5, 0), img_width - 1);
     res[si+1] = min(max(cy - h * 0.5, 0), img_height - 1);
     res[si+2] = max(min(cx + w * 0.5, img_width - 1), 0);
     res[si+3] = max(min(cy + h * 0.5, img_height - 1), 0);
}

__global__ void pickElementsKernel(float *src, float *dst, int *idx, int stride, int block_size, int total)
{
     int di = blockIdx.x * block_size + threadIdx.x;
     if (di >= total)
          return;
     int si = idx[di];
     for (int i = 0; i < stride; i++)
          dst[di*stride+i] = src[si*stride+i];
}

static void assertTensor(const Tensor *tensor)
{
     assert(tensor && tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0);
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isTensorValid(const Tensor *tensor)
{
     assert(tensor);
     assert(tensor->data);
     assert(tensor->ndim < MAXDIM && tensor->ndim > 0 );
     assert(tensor->len == computeLength(tensor->ndim, tensor->dims));
     return (tensor && tensor->data &&
             tensor->ndim < MAXDIM && tensor->ndim > 0 &&
             tensor->len == computeLength(tensor->ndim, tensor->dims));
}

int isShapeEqual(const Tensor *t1, const Tensor *t2)
{
     assertTensor(t1);
     assertTensor(t2);
     if (t1->ndim == t2->ndim) {
          int ndim = t1->ndim;
          while (--ndim >= 0)
               if (t1->dims[ndim] != t2->dims[ndim])
                    return 0;
          return 1;
     }
     return 0;
}

/* can only identify host memory alloced by hipHostMalloc, etc */
int isHostMem(const void *ptr)
{
     hipPointerAttribute_t attributes;
     checkError(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeHost;
}

int isDeviceMem(const void *ptr)
{
     hipPointerAttribute_t attributes;
     checkError(hipPointerGetAttributes(&attributes, ptr));
     return attributes.memoryType == hipMemoryTypeDevice;
}

void *cloneMem(const void *src, size_t size, CloneKind kind)
{
     assert(src);
     void *p;
     switch (kind) {
     case H2H:
          p = sdt_alloc(size);
          memmove(p, src, size);
          return p;
     case H2D:
          checkError(hipMalloc(&p, size));
          checkError(hipMemcpy(p, src, size, hipMemcpyHostToDevice));
          return p;
     case D2D:
          assert(isDeviceMem(src));
          checkError(hipMalloc(&p, size));
          checkError(hipMemcpy(p, src, size, hipMemcpyDeviceToDevice));
          return p;
     case D2H:
          assert(isDeviceMem(src));
          p = sdt_alloc(size);
          checkError(hipMemcpy(p, src, size, hipMemcpyDeviceToHost));
          return p;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }

}

Tensor *cloneTensor(const Tensor *src, CloneKind kind)
{
     assert(isTensorValid(src));
     float *data = (float *)cloneMem(src->data, src->len * sizeof(float), kind);
     Tensor *dst = createTensor(data, src->ndim, src->dims);
     return dst;
}

void *repeatMem(void *data, size_t size, int times, CloneKind kind)
{
     assert(data && times > 0);
     void *p, *dst;
     int i;
     switch (kind) {
     case H2H:
          dst = p = sdt_alloc(size * times);
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               memmove(p, data, size);
          return dst;
     case H2D:
          checkError(hipMalloc(&p, size * times));
          dst = p;
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyHostToDevice));
          return dst;
     case D2D:
          assert(isDeviceMem(data));
          checkError(hipMalloc(&p, size * times));
          dst = p;
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyDeviceToDevice));
          return dst;
     case D2H:
          assert(isDeviceMem(data));
          dst = p = sdt_alloc(size * times);
          for (i = 0; i < times; i++, p = (char *)p + size * times)
               checkError(hipMemcpy(p, data, size, hipMemcpyDeviceToHost));
          return dst;
     default:
          fprintf(stderr, "unknown CloneKind %d\n", kind);
          return NULL;
     }
}


int computeLength(int ndim, const int *dims)
{
     if (dims) {
          int i, len = 1;
          for (i = 0; i < ndim; i++)
               len *= dims[i];
          return len;
     }
     fprintf(stderr, "Warning: null dims in computeLength\n");
     return 0;
}

Tensor *createTensor(float *data, int ndim, const int *dims)
{
     Tensor *t = (Tensor *)sdt_alloc(sizeof(Tensor));
     t->data = data;
     t->ndim = ndim;
     t->dims = (int *)sdt_alloc(sizeof(int) * ndim);
     memmove(t->dims, dims, sizeof(int) * ndim);
     t->len = computeLength(ndim, dims);
     return t;
}

Tensor *mallocTensor(int ndim, const int* dims, const MallocKind mkind)
{
     Tensor *t = createTensor(NULL, ndim, dims);
     float *f;

     switch (mkind) {
     case HOST:
          f = (float *)sdt_alloc(t->len * sizeof(float));
          break;
     case DEVICE:
          checkError(hipMalloc(&f, t->len * sizeof(float)));
          break;
     default:
          fprintf(stderr, "unknown MallocKind %d\n", mkind);
          return NULL;
     }

     t->data = f;
     return t;
}

void freeTensor(Tensor *t, int do_free_data)
{
     assert(isTensorValid(t));
     sdt_free(t->dims);
     if (do_free_data) {
          if (isDeviceMem(t->data))
               checkError(hipFree(t->data));
          else
               sdt_free(t->data);
     }
     sdt_free(t);
}

void fprintShape(FILE *stream, const Tensor *tensor)
{
     assert(tensor && tensor->dims);

     int i;
     fprintf(stream, "(");
     for (i = 0; i < tensor->ndim; i++) {
          fprintf(stream, "%d", tensor->dims[i]);
          if (i != tensor->ndim-1)
               fprintf(stream, ", ");
     }
     fprintf(stream, ")\n");
}

void fprintTensor(FILE *stream, const Tensor *tensor, const char *fmt)
{
     assertTensor(tensor);
     int dim_sizes[MAXDIM], dim_levels[MAXDIM]; /* dimision size and how deep current chars go */
     int ndim = tensor->ndim, len = tensor->len, *dims = tensor->dims; /* pointer short cut */
     float *data = tensor->data;
     char left_buf[MAXDIM+1], right_buf[MAXDIM+1]; /* buffer for brackets */
     char *lp = left_buf, *rp = right_buf;
     size_t right_len;
     int i, j, k;

     dim_sizes[ndim-1] = tensor->dims[ndim-1];
     dim_levels[ndim-1] = 0;
     for (i = ndim-2; i >= 0; i--) {
          dim_sizes[i] = dims[i] * dim_sizes[i+1];
          dim_levels[i] = 0;
     }
     for (i = 0; i < len; i++) {
          for (j = 0; j < ndim; j++) {
               if (i % dim_sizes[j] == 0)
                    dim_levels[j]++;
               if (dim_levels[j] == 1) {
                    *lp++ = '[';
                    dim_levels[j]++;
               }
               if (dim_levels[j] == 3) {
                    *rp++ = ']';
                    if (j != 0 && dim_levels[j] > dim_levels[j-1]) {
                         *lp++ = '[';
                         dim_levels[j] = 2;
                    } else
                         dim_levels[j] = 0;
               }
          }
          *lp = *rp = '\0';
          fprintf(stream, "%s", right_buf);
          if (*right_buf != '\0') {
               fprintf(stream, "\n");
               right_len = strlen(right_buf);
               for (k = ndim-right_len; k > 0; k--)
                    fprintf(stream, " ");
          }
          fprintf(stream, "%s", left_buf);
          if (*left_buf == '\0')
               fprintf(stream, " ");
          fprintf(stream, fmt, data[i]);
          lp = left_buf, rp = right_buf;
     }
     for (j = 0; j < ndim; j++)
          fprintf(stream, "]");
     fprintf(stream, "\n");
}

void printTensor(const Tensor *tensor, const char *fmt)
{
     fprintTensor(stdout, tensor, fmt);
}

void fprintDeviceTensor(FILE *stream, const Tensor *d_tensor, const char *fmt)
{
     assert(isTensorValid(d_tensor));
     Tensor *h_tensor = cloneTensor(d_tensor, D2H);
     fprintTensor(stream, h_tensor, fmt);
     free(h_tensor->data); /* TODO: free t_tensor */
}

void printDeviceTensor(const Tensor *d_tensor, const char *fmt)
{
     fprintDeviceTensor(stdout, d_tensor, fmt);
}

void saveTensor(const char *file_name, const Tensor *tensor, const char *fmt)
{
     FILE *fp = fopen(file_name, "w");
     fprintTensor(fp, tensor, fmt);
     fclose(fp);
}

void saveDeviceTensor(const char *file_name, const Tensor *d_tensor, const char *fmt)
{
     FILE *fp = fopen(file_name, "w");
     fprintDeviceTensor(fp, d_tensor, fmt);
     fclose(fp);
}

/* Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src)); */
/*      assert(dim <= src->ndim && dim >= 0); */
/*      assert(len+start <= src->dims[dim]); */

/*      Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /\* new tensor *\/ */
/*      dst->ndim = src->ndim; */
/*      dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim); */
/*      memmove(dst->dims, src->dims, sizeof(int) * dst->ndim); */
/*      dst->dims[dim] = len; */
/*      dst->len = src->len / src->dims[dim] * len; */
/*      dst->data = (float *)sdt_alloc(dst->len * sizeof(float)); */
/*      return dst; */
/* } */

/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i, block_size, block_num; /\* block size and number for copy operation *\/ */
/*      for (i = dim+1, block_size = 1; i < dst->ndim; i++) */
/*           block_size *= dst->dims[i]; */
/*      for (i = 0, block_num = 1; i <= dim; i++) */
/*           block_num *= dst->dims[i]; */

/*      int index; */
/*      float *dp = dst->data, *sp = src->data; */
/*      size_t floats_size = block_size * sizeof(float); */
/*      for (i = 0; i < block_num; i++) { */
/*           index = i / len * src->dims[dim] + i % len + start; */
/*           memmove(dp+i*block_size, sp+index*block_size, floats_size); */
/*      } */

/*      return dst; */
/* } */

Tensor *createSlicedTensor(const Tensor *src, int dim, int start, int len)
{
     assert(isTensorValid(src));
     assert(dim <= MAXDIM);
     assert(len+start <= src->dims[dim]);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor)); /* new tensor */
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = len;
     dst->len = src->len / src->dims[dim] * len;
     checkError(hipMalloc(&dst->data, sizeof(float) * dst->len));
     return dst;
}

/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(isDeviceMem(src->data) && isDeviceMem(dst->data)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i, block_size, block_num; /\* block size and number of cuda threads *\/ */
/*      int ddim = dst->dims[dim], sdim = src->dims[dim]; */
/*      for (i = dim+1, block_size = 1; i < dst->ndim; i++) */
/*           block_size *= dst->dims[i]; */
/*      for (i = 0, block_num = 1; i <= dim; i++) */
/*           block_num *= dst->dims[i]; */

/*      sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, sdim, ddim, start, block_size); */
/*      return dst; */
/* } */
/* axes should be in ascending order */
/* Tensor *sliceTensor(const Tensor *src, Tensor *dst, int naxis, int *axes, int *starts, int *ends, int *strides, int **workspace) */
/* { */
/*      assert(isTensorValid(src) && isTensorValid(dst)); */
/*      assert(isDeviceMem(src->data) && isDeviceMem(dst->data)); */
/*      assert(dst->ndim == src->ndim); */
/*      for (int i = 0; i < dst->ndim; i++) */
/*           assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]); */

/*      int i; */
/*      int *sdims, *ddims, *axes_device, *n_pick_ids_host, **pick_ids_host, *n_pick_ids_device, **pick_ids_device, *sids, *dids, *ends_tmp; */
/*      sdims = (int *)cloneMem(src->dims, sizeof(int)*src->ndim, H2D); */
/*      ddims = (int *)cloneMem(dst->dims, sizeof(int)*dst->ndim, H2D); */
/*      axes_device = (int *)cloneMem(axes, sizeof(int)*naxis, H2D); */
/*      n */
/*      ends_tmp = (int *)cloneMem(ends, sizeof(int)*naxis, H2H); */
/*      if (!workspace) { */
/*           checkError(hipMalloc(&sids, sizeof(int) * dst->ndim * dst->len)); */
/*           checkError(hipMalloc(&dids, sizeof(int) * dst->ndim * dst->len)); */
/*      } else { */
/*           sids = workspace[0]; */
/*           dids = workspace[1]; */
/*      } */
/*      for (i = 0; i < naxis; i++) { */
/*           if (ends_tmp[i] > src->dims[axes[i]]) */
/*                ends_tmp[i] = src->dims[axes[i]]; */
/*           n_pick_ids[i] = */
/*      } */

/*      int thread_num, block_size, block_num; /\* block size and number of cuda threads *\/ */
/*      thread_num = dst->len; */
/*      block_size = MAX_THREADS_PER_BLOCK; */
/*      block_num = thread_num / block_size + 1; */

/*      sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, start, s_vol, d_vol, vol, block_size, thread_num); */

/*      checkError(hipFree(sdims)); */
/*      checkError(hipFree(ddims)); */
/*      checkError(hipFree(axes_device)); */
/*      checkError(hipFree(n_pick_ids)); */
/*      sdt_free(ends_tmp); */
/*      return dst; */
/* } */

Tensor *sliceTensor(const Tensor *src, Tensor *dst, int dim, int start, int len)
{
     assert(isTensorValid(src));
     assert(isTensorValid(dst));
     assert(isDeviceMem(src->data) && isDeviceMem(dst->data));
     assert(dst->ndim == src->ndim);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == len : dst->dims[i] == src->dims[i]);

     int i, d_vol, s_vol, vol;
     int thread_num, block_size, block_num; /* block size and number of cuda threads */
     for (i = dim+1, vol = 1; i < dst->ndim; i++)
          vol *= dst->dims[i];
     d_vol = vol * dst->dims[dim];
     s_vol = vol * src->dims[dim];
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     sliceTensorKernel<<<block_num, block_size>>>(src->data, dst->data, start, s_vol, d_vol, vol, block_size, thread_num);
     return dst;
}

/* in-place reshape tensor */
Tensor *reshapeTensor(const Tensor *src, int newNdim, const int *newDims)
{
     assert(isTensorValid(src));
     assert(newDims);
     /* fprintf(stderr, "len = %d, compute = %d\n", src->len, computeLength(newNdim, newDims)); */
     assert(src->len == computeLength(newNdim, newDims));
     Tensor *dst = createTensor(src->data, newNdim, newDims); /* new tensor */
     return dst;
}

Tensor *createReducedTensor(const Tensor *src, int dim)
{
     assert(isTensorValid(src));
     assert(dim < src->ndim && dim >= 0);

     Tensor *dst = (Tensor *)sdt_alloc(sizeof(Tensor));
     dst->ndim = src->ndim;
     dst->dims = (int *)sdt_alloc(sizeof(int) * dst->ndim);
     memmove(dst->dims, src->dims, sizeof(int) * dst->ndim);
     dst->dims[dim] = 1;
     dst->len = computeLength(dst->ndim, dst->dims);
     checkError(hipMalloc(&dst->data, sizeof(float) * dst->len));
     return dst;
}

void *reduceArgMax(const Tensor *src, Tensor *dst, Tensor *arg, int dim)
{
     assert(isTensorValid(src) && isTensorValid(dst) && isTensorValid(arg));
     assert(isDeviceMem(src->data) && isDeviceMem(dst->data) && isDeviceMem(arg->data));
     assert(dim < src->ndim && dim >= 0);
     for (int i = 0; i < dst->ndim; i++)
          assert(i == dim ? dst->dims[i] == 1 : dst->dims[i] == src->dims[i] &&
                 i == dim ? arg->dims[i] == 1 : arg->dims[i] == src->dims[i]);

     /* suppose the shape of src is [N, C, H, W], dim = 1, then thread_num is N x H x W
        reduce_vol is H x W, index_vol is C x H x W */
     int i, thread_num, block_size, block_num, reduce_vol, index_vol;
     for (i = dim+1, thread_num = 1; i < dst->ndim; i++)
          thread_num *= dst->dims[i];
     reduce_vol = thread_num;
     index_vol = thread_num * src->dims[dim];
     for (i = 0; i < dim; i++)
          thread_num *= dst->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     reduceArgMaxKernel<<<block_num, block_size>>>(src->data, dst->data, arg->data, src->dims[dim], reduce_vol, index_vol, block_size, thread_num);
     return dst;
}

Tensor *multiplyElement(const Tensor *src1, const Tensor *src2, Tensor *dst)
{
     assert(isShapeEqual(src1, src2));
     assert(isShapeEqual(src1, dst));
     assert(isDeviceMem(src1->data) && isDeviceMem(src2->data) && isDeviceMem(dst->data));

     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     multiplyElementKernel<<<block_num, block_size>>>(src1->data, src2->data, dst->data, block_size, dst->len);
     return dst;
}

/* (optional) workspace size equals (sizeof(int) * dst->ndim * dst->len), two of them */
Tensor *transposeTensor(const Tensor *src, Tensor *dst, int *axes, int **workspace)
{
     assert(isTensorValid(src) && isTensorValid(dst));
     assert(src->len == dst->len);
     assert(src->ndim == dst->ndim);

     int *s_ids, *d_ids, *s_dims, *d_dims;
     int thread_num, block_size, block_num;
     thread_num = dst->len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;
     s_dims = (int *)cloneMem(src->dims, sizeof(int) * src->ndim, H2D);
     d_dims = (int *)cloneMem(dst->dims, sizeof(int) * dst->ndim, H2D);
     if (!workspace) {
          checkError(hipMalloc(&s_ids, sizeof(int) * dst->ndim * thread_num));
          checkError(hipMalloc(&d_ids, sizeof(int) * dst->ndim * thread_num));
     } else {
          s_ids = workspace[0];
          d_ids = workspace[1];
     }

     transposeTensorKernel<<<block_num, block_size>>>(src->data, dst->data, dst->ndim, s_dims, d_dims, s_ids, d_ids, axes, block_size, thread_num);

     if (!workspace) {
          checkError(hipFree(s_ids));
          checkError(hipFree(d_ids));
     }
     checkError(hipFree(s_dims));
     checkError(hipFree(d_dims));
     return dst;
}

/* transform from bbox delta to bbox coordinates, using hyper param EXP_THRESH = 1.0.
   delta, anchor, res are all of the same shape [..., 4]
   width and height are resized image width and height.
   x_scales and y_scales are (temporary) pointers to width/original_width and height/original_height. */
Tensor *transformBboxSQD(const Tensor *delta, const Tensor *anchor, Tensor *res, float width, float height, float img_width, float img_height, int x_shift, int y_shift)
{
     assert(isShapeEqual(delta, anchor));
     assert(isShapeEqual(delta, res));
     assert(delta->ndim == 5);
     assert(delta->dims[4] == 4);
     assert(isDeviceMem(delta->data) && isDeviceMem(anchor->data) && isDeviceMem(res->data));

     /* take 4 elements from each of delta and anchor,
        and put 4 result elements to res in one thread */
     int i, thread_num, block_size, block_num;
     for (i = 0, thread_num = 1; i < res->ndim-1; i++)
          thread_num *= res->dims[i];
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     transformBboxSQDKernel<<<block_num, block_size>>>(delta->data, anchor->data, res->data, width, height, img_width, img_height, x_shift, y_shift, block_size, thread_num);
     return res;
}

void tensorIndexSort(Tensor *src, int *idx)
{
     assert(isTensorValid(src));
     assert(idx);
     assert(isDeviceMem(src->data) && isDeviceMem(idx));

     /* the thrust call below can be unreliable, sometimes produces error */
     /* now it works with compilation flag -arch=sm_35 */
     /* TODO: replace thrust call by our own kernel */
     thrust::sort_by_key(thrust::device, src->data, src->data + src->len, idx, thrust::greater<float>());
}

void pickElements(float *src, float *dst, int stride, int *idx, int len)
{
     assert(src && dst && idx);
     assert(isDeviceMem(src) && isDeviceMem(dst) && isDeviceMem(idx));

     int thread_num, block_size, block_num;
     thread_num = len;
     block_size = MAX_THREADS_PER_BLOCK;
     block_num = thread_num / block_size + 1;

     pickElementsKernel<<<block_num, block_size>>>(src, dst, idx, stride, block_size, thread_num);
}

/* void pickElements(float* src,float* dst,int stride,int* idx,int len) */
/* { */
/*      assert(src && dst && idx); */

/*      for (int i = 0; i < len; i++) { */
/*           for (int j = 0; j < stride; j++) { */
/*                fprintf(stderr, "i: %d j: %d idx[i]: %d src[idx[i]]: %.2f", */
/*                        i, j, idx[i], src[idx[i]]); */
/*                fprintf(stderr, "\n"); */
/*                dst[i*stride+j] = src[idx[i]*stride+j]; */
/*           } */
/*      } */
/* } */

/* compute the iou of two bboxes whose elements are {top_left_x, top_left_y, bottom_right_x, bottom_right_y} */
float computeIou(float *bbox0, float *bbox1)
{
     assert(bbox0 && bbox1);

     float lr, tb;              /* left-right, top-bottom for intersection*/
     float intersection, total;
     lr = min(bbox0[2], bbox1[2]) - max(bbox0[0], bbox1[0]);
     if (lr >= 0) {
          tb = min(bbox0[3], bbox1[3]) - max(bbox0[1], bbox1[1]);
          if (tb >= 0) {
               intersection = tb * lr + EPSILON;
               total = (bbox0[2] - bbox0[0]) * (bbox0[3] - bbox0[1]) +
                    (bbox1[2] - bbox1[0]) * (bbox1[3] - bbox1[1]) - intersection;
               return intersection / (total + EPSILON);
          }
     }
     return 0;
}
